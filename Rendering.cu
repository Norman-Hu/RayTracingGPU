#include "hip/hip_runtime.h"
#include <Rendering.cuh>
#include <surface_indirect_functions.h>
#include <cstdio>


__global__ void render(Scene * scene, unsigned int w, unsigned int h, float camNear, Vec3 camPos, Matrix4x4 rayTransform, hipSurfaceObject_t surface, hiprandState_t * randState)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x<w && y<h)
	{
		float ndc_x = (2.f*(float)x / (float)w)-1.f;
		float ndc_y = 1.f - (2.f*(float)y / (float)h);
		Vec4 vec{ndc_x, ndc_y, camNear, 1.f};
		vec = rayTransform*vec;
		Vec3 dir = Vec3{vec[0], vec[1], vec[2]};
		dir.normalize();

		Ray ray{camPos, dir.normalized()};

		uchar4 val;
		Vec3 colorVal = Vec3(0.0f, 0.0f, 0.0f);
		int lightsCount = scene->lightCount;

		Vec3 color(0.0f, 0.0f, 0.0f);
		constexpr int maxBounces = 10;

		for (int i=0; i<maxBounces; ++i)
		{
			Hit hitInfo;
			bool hit = scene->hit(ray, 0.001f, 50.0f, hitInfo);
			if (hit)
			{
				BlinnPhongMaterial & mat = scene->materials[hitInfo.materialId];
				if (mat.mirror > 0.f)
					ray = {hitInfo.p, Vec3::reflect(ray.direction, hitInfo.normal)};
				else
				if (mat.refraction)
				{
					Vec3 incident = ray.direction;
					Vec3 normal;
					float ratio;
					if (Vec3::dot(ray.direction, hitInfo.normal) > 0.0f)
					{
						normal = -1.f * hitInfo.normal;
						ratio = mat.refractiveIndex;
					}
					else
					{
						ratio = 1.f / mat.refractiveIndex;
						normal = hitInfo.normal;
					}
					ray = {hitInfo.p, Vec3::refract(incident, normal, ratio).normalized()};
				}
				else
				{
					int lightHits = 0;
					for (int l=0; l<lightsCount; l++)
					{
						Light * light = scene->lights[l];
						LightSamples samples = light->getSamples(randState);

						int sampleHits = 0;

						Vec3 colorForLight(0.0f, 0.0f, 0.0f);
						for (int lightSample=0; lightSample<samples.size; ++lightSample)
						{
							Vec3 lightPos = samples.samples[lightSample];

							Vec3 lightDir = (lightPos - hitInfo.p);
							float lightDistance = lightDir.length();
							lightDir.normalize();

							// check if obstructed
							Hit _unused;
							if (scene->hit({hitInfo.p, lightDir}, 0.001f, lightDistance, _unused))
								continue;

							++sampleHits;

							float diff = max(Vec3::dot(hitInfo.normal, lightDir), 0.0f);
							Vec3 diffuse = mat.diffuse * diff;

							Vec3 viewDir = (-ray.direction).normalized();
							Vec3 halfDir = (lightDir + viewDir).normalized();
							float spec = powf(max(Vec3::dot(hitInfo.normal, halfDir), 0.0f), mat.shininess);
							Vec3 specular = spec * mat.specular;

							Vec3 res = mat.ambient + diffuse + specular;
							colorForLight += res.mulComp(light->color);
						}
						if (sampleHits > 0)
						{
							colorForLight/=sampleHits;
							++lightHits;
							color += colorForLight;
						}
					}
					if (lightHits > 0)
						color /= lightHits;
					break;
				}
			}
			else break;
		}

		float maxVal = max(color.x, max(color.y, color.z));
		if (maxVal > 1.0f)
			color /= maxVal;
		color *= 255.0f;
		val.x = color.x;
		val.y = color.y;
		val.z = color.z;
		val.w = 255;
		// the error on the next line is a lie
		surf2Dwrite<uchar4>(val, surface, (int)sizeof(uchar4)*x, y, hipBoundaryModeClamp);
	}
}

__global__ void setupRandomState(hiprandState_t * state, uint64_t seed)
{
	int tid = gridDim.x*blockDim.x*(blockDim.y*blockIdx.y+threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

__device__ float randUniform(hiprandState_t * state)
{
	int tid = gridDim.x*blockDim.x*(blockDim.y*blockIdx.y+threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	return hiprand_uniform(&state[tid]);
}

__device__ Vec3 randomInSphere(hiprandState_t * state)
{
	constexpr float pi = 3.14159265358979323846f;
	float pitch = randUniform(state)*pi*2.f;
	float yaw = randUniform(state)*pi*2.f;
	Vec3 res;
	res.x = cosf(yaw) * cosf(pitch);
	res.y = sinf(pitch);
	res.z = sinf(yaw) * cosf(pitch);
	res.normalize();
	float distance = sqrtf(randUniform(state));
	return res*distance;
}

__global__ void testFillFramebuffer(unsigned int w, unsigned int h, hipSurfaceObject_t surface)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x<w && y<h)
	{
		uchar4 val = {255, 255, 0, 255};
		surf2Dwrite<uchar4>(val, surface, (int)sizeof(uchar4)*x, y, hipBoundaryModeClamp);
	}
}

__global__ void renderStraight(Scene * scene, unsigned int w, unsigned int h, float camNear, Vec3 camPos, Matrix4x4 rayTransform, hipSurfaceObject_t surface)
{
	float x = w/2.0f;
	float y = h/2.0f;
	float ndc_x = (2.f*(float)x / (float)w)-1.f;
	float ndc_y = 1.f - (2.f*(float)y / (float)h);
	Vec4 vec{ndc_x, ndc_y, camNear, 1.f};
	vec = rayTransform*vec;
	Vec3 dir = Vec3{vec[0], vec[1], vec[2]};
	dir.normalize();

	Ray ray{camPos, dir.normalized()};

	Hit out;
	uchar4 val;
	if (scene->hit(ray, 0.1f, 50.0f, out))
	{
		val = {255, 0, 0, 255};
	}
}