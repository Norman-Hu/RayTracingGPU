#include "hip/hip_runtime.h"
#include <Rendering.cuh>
#include <surface_indirect_functions.h>
#include <cstdio>

#define PI 3.14159265358979323846f
#define INV_PI 0.31830988618379067154f
#define INV_2PI 0.15915494309189533577f

// brdf
__host__ __device__ float distributionGGX(const Vec3 & N, const Vec3 & H, float roughness)
{
    float a      = roughness*roughness;
    float a2     = a*a;
    float NdotH  = fmaxf(Vec3::dot(N, H), 0.0);
    float NdotH2 = NdotH*NdotH;

    float num   = a2;
    float denom = (NdotH2 * (a2 - 1.0) + 1.0);
    denom = PI * denom * denom;

    return num / denom;
}

__host__ __device__ float geometrySchlickGGX(float NdotV, float roughness)
{
    float r = (roughness + 1.0f);
    float k = (r*r) / 8.0f;

    float num   = NdotV;
    float denom = NdotV * (1.0f - k) + k;

    return num / denom;
}

__host__ __device__ float geometrySmith(const Vec3 & N, const Vec3 & V, const Vec3 & L, float roughness)
{
    float NdotV = fmaxf(Vec3::dot(N, V), 0.0f);
    float NdotL = fmaxf(Vec3::dot(N, L), 0.0f);
    float ggx2  = geometrySchlickGGX(NdotV, roughness);
    float ggx1  = geometrySchlickGGX(NdotL, roughness);

    return ggx1 * ggx2;
}

__host__ __device__ Vec3 fresnelSchlick(float cosTheta, const Vec3 & F0)
{
    float tmp = 1.0f - cosTheta;
    tmp = tmp < 0.0f ? 0.0f : tmp;
    tmp = tmp > 1.0f ? 1.0f : tmp;
    return F0 + Vec3{1.0f-F0.x, 1.0f-F0.y, 1.0f-F0.z} * powf(tmp, 5.0f);
}

__host__ __device__ Vec3 sampleUniformHemisphere(float u, float v)
{
    float z = u;
    float r = sqrtf(1.0f - z*z);
    float phi = 2.0f * PI * v;
    return {r * cosf(phi), r * sinf(phi), z};
}

__host__ __device__ Vec3 sampleUniformSphere(float u, float v)
{
    float z = 1.0f - 2.0f * u;
    float r = sqrtf(1.0f - z*z);
    float phi = 2.0f * PI * v;
    return {r * cosf(phi), r * sinf(phi), z};
}


__global__ void render(Scene * scene, unsigned int w, unsigned int h, float camNear, Vec3 camPos, Matrix4x4 rayTransform, hipSurfaceObject_t surface, hiprandState_t * randState, bool sampleLights)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = gridDim.x*blockDim.x*(blockDim.y*blockIdx.y+threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (x<w && y<h)
	{
        // Find ray
		float ndc_x = (2.f*(float)x / (float)w)-1.f;
		float ndc_y = 1.f - (2.f*(float)y / (float)h);
		Vec4 vec{ndc_x, ndc_y, camNear, 1.f};
		vec = rayTransform*vec;
		Vec3 dir = Vec3{vec[0], vec[1], vec[2]};
		dir.normalize();
		Ray ray{camPos, dir.normalized()};

        // init
        Vec3 L{0.0f, 0.0f, 0.0f};
        Vec3 beta{1.0f, 1.0f, 1.0f};
		constexpr int maxBounces = 5;
        bool specularBounce = true;
        constexpr int maxDepth = 5;
        int depth = 0;

		while (beta.max() > 0.0f)
        {
            Hit hitInfo;
            bool hit = scene->hit(ray, 0.001f, 100.0f, hitInfo);
            if (!hit) {
                // infinite lights, which have a fixed color
                if (!sampleLights || specularBounce)
                    L += compwise_mul(beta, {.2f, .2f, .2f});
                break;
            }

            // Get material
            PBRMaterial &mat = scene->materials[hitInfo.materialId];

            // if we don't sample lights directly, account for emissive surfaces
            if (!sampleLights || specularBounce)
            {
                L += compwise_mul(beta, mat.emissive);
            }

            // end path if max depth is reached
            if (depth++ == maxDepth)
                break;

            // direct illumination
            Vec3 wo = -ray.direction;
            if (sampleLights)
            {

            }

            if (false) // TODO: Importance sampling
            {

            }
            else
            {
                // Uniform sampling
                Vec3 wi;
                float pdf = INV_2PI;
                if (false) // TODO: reflective+transmissive
                {

                }
                else
                {
                    wi = sampleUniformHemisphere(hiprand_uniform(randState+tid), hiprand_uniform(randState+tid));
                    pdf = INV_2PI;
                    // reflection
                    if (Vec3::dot(wo, hitInfo.normal) * Vec3::dot(wi, hitInfo.normal) < 0)
                        wi = -wi;
                    // transmission
//                    else if (Vec3::dot(wo, hitInfo.normal) * Vec3::dot(wi, hitInfo.normal) > 0)
//                        wi = -wi;
                }

                Vec3 H = (wo+wi); H.normalize();
                Vec3 F0 = Vec3{0.04f, 0.04f, 0.04f};
                F0      = Vec3::mix(F0, mat.albedo, mat.metallic);
                Vec3 F  = fresnelSchlick(fmaxf(Vec3::dot(H, wo), 0.0), F0);
                float NDF = distributionGGX(hitInfo.normal, H, mat.roughness);
                float G   = geometrySmith(hitInfo.normal, wo, wi, mat.roughness);
                Vec3 numerator    = NDF * G * F;
                float denominator = 4.0f * fmaxf(Vec3::dot(hitInfo.normal, wo), 0.0f) * fmaxf(Vec3::dot(hitInfo.normal, wi), 0.0f)  + 0.0001f;
                Vec3 specular     = numerator / denominator;
                Vec3 kS = F;
                Vec3 kD = Vec3{1.0f, 1.0f, 1.0f} - kS;
                kD *= 1.0f - mat.metallic;

                beta = compwise_mul(specular * fabsf(Vec3::dot(wi, hitInfo.normal)) / pdf, beta);
                specularBounce = false;
                ray.origin = hitInfo.p;
                ray.direction = wi;
            }

//			if (hit)
//			{
//				int lightHits = 0;
//				for (int l=0; l<lightsCount; l++)
//				{
//					Light * light = scene->lights[l];
//					LightSamples samples = light->getSamples(randState);
//
//					int sampleHits = 0;
//
//					Vec3 colorForLight(0.0f, 0.0f, 0.0f);
//					for (int lightSample=0; lightSample<samples.size; ++lightSample)
//					{
//						Vec3 lightPos = samples.samples[lightSample];
//
//						Vec3 lightDir = (lightPos - hitInfo.p);
//						float lightDistance = lightDir.length();
//						lightDir.normalize();
//
//						// check if obstructed
//						Hit _unused;
//						if (scene->hit({hitInfo.p, lightDir}, 0.001f, lightDistance, _unused))
//							continue;
//
//						++sampleHits;
//
////						float diff = max(Vec3::dot(hitInfo.normal, lightDir), 0.0f);
////						Vec3 diffuse = mat.diffuse * diff;
////
////						Vec3 viewDir = (-ray.direction).normalized();
////						Vec3 halfDir = (lightDir + viewDir).normalized();
////						float spec = powf(max(Vec3::dot(hitInfo.normal, halfDir), 0.0f), mat.shininess);
////						Vec3 specular = spec * mat.specular;
////
////						Vec3 res = mat.ambient + diffuse + specular;
////						colorForLight += res.mulComp(light->color);
//						colorForLight += mat.albedo;
//					}
//					if (sampleHits > 0)
//					{
//						colorForLight/=sampleHits;
//						++lightHits;
//						color += colorForLight;
//					}
//				}
//				if (lightHits > 0)
//					color /= lightHits;
//				break;
//			}
//			else break;
//		}
//
//		float maxVal = max(color.x, max(color.y, color.z));
//		if (maxVal > 1.0f)
//			color /= maxVal;
//		color *= 255.0f;
//		val.x = color.x;
//		val.y = color.y;
//		val.z = color.z;
//		val.w = 255;
//		surf2Dwrite<uchar4>(val, surface, (int)sizeof(uchar4)*x, y, hipBoundaryModeClamp);
        }


        L = compwise_div(L, (L + Vec3(1.0f, 1.0f, 1.0f)));
        L = compwise_pow(L, Vec3(1.0f/2.2f, 1.0f/2.2f, 1.0f/2.2f));


        uchar4 val;
        val.x = static_cast<unsigned char>(L.x*255.0f);
        val.y = static_cast<unsigned char>(L.y*255.0f);
        val.z = static_cast<unsigned char>(L.z*255.0f);
        val.w = 255;
		surf2Dwrite<uchar4>(val, surface, (int)sizeof(uchar4)*x, y, hipBoundaryModeClamp);
	}
}

__global__ void setupRandomState(hiprandState_t * state, uint64_t seed)
{
	int tid = gridDim.x*blockDim.x*(blockDim.y*blockIdx.y+threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

__device__ float randUniform(hiprandState_t * state)
{
	int tid = gridDim.x*blockDim.x*(blockDim.y*blockIdx.y+threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	return hiprand_uniform(&state[tid]);
}

__device__ Vec3 randomInSphere(hiprandState_t * state)
{
	constexpr float pi = 3.14159265358979323846f;
	float pitch = randUniform(state)*pi*2.f;
	float yaw = randUniform(state)*pi*2.f;
	Vec3 res;
	res.x = cosf(yaw) * cosf(pitch);
	res.y = sinf(pitch);
	res.z = sinf(yaw) * cosf(pitch);
	res.normalize();
	float distance = sqrtf(randUniform(state));
	return res*distance;
}

__global__ void testFillFramebuffer(unsigned int w, unsigned int h, hipSurfaceObject_t surface)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x<w && y<h)
	{
		uchar4 val = {255, 255, 0, 255};
		surf2Dwrite<uchar4>(val, surface, (int)sizeof(uchar4)*x, y, hipBoundaryModeClamp);
	}
}

__global__ void renderStraight(Scene * scene, unsigned int w, unsigned int h, float camNear, Vec3 camPos, Matrix4x4 rayTransform, hipSurfaceObject_t surface)
{
	float x = w/2.0f;
	float y = h/2.0f;
	float ndc_x = (2.f*(float)x / (float)w)-1.f;
	float ndc_y = 1.f - (2.f*(float)y / (float)h);
	Vec4 vec{ndc_x, ndc_y, camNear, 1.f};
	vec = rayTransform*vec;
	Vec3 dir = Vec3{vec[0], vec[1], vec[2]};
	dir.normalize();

	Ray ray{camPos, dir.normalized()};

	Hit out;
	uchar4 val;
	if (scene->hit(ray, 0.1f, 50.0f, out))
	{
		val = {255, 0, 0, 255};
	}
}