#include "hip/hip_runtime.h"
#include <Object.cuh>
#include <CudaHelpers.h>


__device__ bool Sphere::hit(const Ray & ray, float tmin, float tmax, Hit & out)
{
	Vec3 centerToOrigin = ray.origin - c;
	float a = ray.direction.sqLength();
	float b = Vec3::dot(centerToOrigin, ray.direction);
	float c = centerToOrigin.sqLength() - r*r;

	float delta = b*b - a*c;
	if (delta < 0)
	{
		return false;
	}
	float sqrtDelta = sqrtf(delta);

	float root = (-b-sqrtDelta/a);
	if (root < tmin || root > tmax)
	{
		root = (-b+sqrtDelta)/a;
		if (root < tmin || root > tmax)
			return false;
	}

	out.t = root;
	out.p = ray.origin+ray.direction*root;
	out.normal = (out.p - this->c)/r;
	out.materialId = materialId;
	return true;
}

__device__ bool Square::hit(const Ray & ray, float tmin, float tmax, Hit & out)
{
    if (Vec3::dot(ray.direction, n) > 0.f)
        return false;

	float t = Vec3::dot(p - ray.origin, n)/Vec3::dot(ray.direction, n);
	if (t < tmin || t > tmax)
		return false;

	Vec3 inter = ray.origin + t * ray.direction;

	Vec3 toIntersection = inter - p;
	float dotRight = Vec3::dot(toIntersection, right);
	float dotUp = Vec3::dot(toIntersection, up);

	if (!(dotRight >= 0 && dotRight <= Vec3::dot(right, right) && dotUp >= 0 && dotUp <= Vec3::dot(up, up)))
		return false;

	out.t = t;
	out.p = inter;
	out.normal = n;
	out.materialId = materialId;

	return true;
}

__device__ bool Mesh::hit(const Ray & ray, float tmin, float tmax, Hit & out)
{
	bool intersects = false;
	for (unsigned int i = 0; i < indices_count; i += 3)
	{
		// Möller–Trumbore intersection algorithm
		// From https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
		constexpr float EPSILON = 1e-7f;
		Vec3 v0 = vertices[indices[i]], v1 = vertices[indices[i + 1]], v2 = vertices[indices[i + 2]];
		Vec3 e1 = v1 - v0;
		Vec3 e2 = v2 - v0;
		Vec3 crossRayE2 = Vec3::cross(ray.direction, e2);
		float det = Vec3::dot(e1, crossRayE2);
		if (det > -EPSILON && det < EPSILON) // parallel
			continue;

		float invDet = 1.0f/det;
		Vec3 s = ray.origin - v0;
		float u = invDet * Vec3::dot(s, crossRayE2);
		if (u < 0.f || u > 1.f)
			continue;

		Vec3 crossSE1 = Vec3::cross(s, e1);
		float v = invDet * Vec3::dot(ray.direction, crossSE1);
		if (v < 0.f || u+v > 1.f)
			continue;

		float t = invDet * Vec3::dot(e2, crossSE1);
		if (t < tmin || t > tmax)
			continue;

		if (!intersects || t < out.t)
		{
			out.t = t;
			out.p = ray.origin + t * ray.direction;
			out.normal = Vec3::cross(e1, e2); // FIXME: Interpolate normals using barycentric coordinates
			out.materialId = materialId;
			intersects = true;
		}
	}
	return intersects;
}

__device__ Mesh::Mesh()
: vertices(nullptr)
, normals(nullptr)
, indices(nullptr)
, vertices_count(0)
, normals_count(0)
, indices_count(0)
{
}

__device__ Mesh::~Mesh()
{
	delete[] vertices;
	delete[] normals;
	delete[] indices;
}

Mesh * createMesh()
{
	Mesh * res;
	Mesh ** ptr_d_Mesh;
	hipMalloc(&ptr_d_Mesh, sizeof(Mesh *));
	d_createMesh<<<1, 1>>>(ptr_d_Mesh);
	hipMemcpy(&res, ptr_d_Mesh, sizeof(Mesh *), hipMemcpyDeviceToHost);
	hipFree(ptr_d_Mesh);
	return res;
}

__global__ void d_createMesh(Mesh ** ptr_d_mesh)
{
	Mesh * mesh = new Mesh();
	*ptr_d_mesh = mesh;
}

void setMeshMaterial(Mesh * mesh, unsigned int index)
{
	d_setMeshMaterial<<<1, 1>>>(mesh, index);
}

__global__ void d_setMeshMaterial(Mesh * mesh, unsigned int index)
{
	mesh->materialId = index;
}

void setMeshVertices(Mesh * mesh, Vec3 * vertices, unsigned int verticesCount)
{
	d_setMeshVerticesCount<<<1, 1>>>(mesh, verticesCount);
	syncAndCheckErrors();
	Vec3 * d_vertices;
	hipMalloc(&d_vertices, sizeof(Vec3)*verticesCount);
	hipMemcpy(d_vertices, vertices, sizeof(Vec3)*verticesCount, hipMemcpyHostToDevice);
	d_copyVertices<<<1, 1>>>(mesh, d_vertices, verticesCount);
	syncAndCheckErrors();
	hipFree(d_vertices);
}

__global__ void d_copyVertices(Mesh * mesh, Vec3 * d_vertices, unsigned int verticesCount)
{
	memcpy((void*)mesh->vertices, (void*)d_vertices, sizeof(Vec3)*verticesCount);
}

__global__ void d_setMeshVerticesCount(Mesh * mesh, unsigned int verticesCount)
{
	delete [] mesh->vertices;
	mesh->vertices = new Vec3[verticesCount];
	mesh->vertices_count = verticesCount;
}

void setMeshIndices(Mesh * mesh, unsigned int * indices, unsigned int indicesCount)
{
	d_setMeshIndicesCount<<<1, 1>>>(mesh, indicesCount);
	syncAndCheckErrors();
	unsigned int * d_indices;
	hipMalloc(&d_indices, sizeof(unsigned int)*indicesCount);
	hipMemcpy(d_indices, indices, sizeof(unsigned int)*indicesCount, hipMemcpyHostToDevice);
	d_copyIndices<<<1, 1>>>(mesh, d_indices, indicesCount);
	syncAndCheckErrors();
	hipFree(d_indices);
}

__global__ void d_copyIndices(Mesh * mesh, unsigned int * d_indices, unsigned int indicesCount)
{
	memcpy((void*)mesh->indices, (void*)d_indices, sizeof(unsigned int)*indicesCount);
}

__global__ void d_setMeshIndicesCount(Mesh * mesh, unsigned int indicesCount)
{
	delete [] mesh->indices;
	mesh->indices = new unsigned int[indicesCount];
	mesh->indices_count = indicesCount;
}

void setMeshNormals(Mesh * mesh, Vec3 * normals, unsigned int normalsCount)
{
	d_setMeshNormalsCount<<<1, 1>>>(mesh, normalsCount);
	syncAndCheckErrors();
	Vec3 * d_normals;
	hipMalloc(&d_normals, sizeof(Vec3)*normalsCount);
	hipMemcpy(d_normals, normals, sizeof(Vec3)*normalsCount, hipMemcpyHostToDevice);
	d_copyNormals<<<1, 1>>>(mesh, d_normals, normalsCount);
	syncAndCheckErrors();
	hipFree(d_normals);
}

__global__ void d_copyNormals(Mesh * mesh, Vec3 * d_normals, unsigned int normalsCount)
{
	memcpy((void*)mesh->normals, (void*)d_normals, sizeof(Vec3)*normalsCount);
}

__global__ void d_setMeshNormalsCount(Mesh * mesh, unsigned int normalsCount)
{
	delete [] mesh->normals;
	mesh->normals = new Vec3[normalsCount];
	mesh->normals_count = normalsCount;
}
