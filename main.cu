#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <Matrix.cuh>
#include <Camera.h>
#include <Scene.cuh>
#include <Rendering.cuh>
#include <iostream>
#include <InputHandler.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <CudaHelpers.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <Importer.cuh>


void destroyBuffers(unsigned int rb, unsigned int fb)
{
	glDeleteRenderbuffers(1, &rb);
	glDeleteFramebuffers(1, &fb);
}

// from https://gist.github.com/allanmac/4ff11985c3562830989f
void setTitleFPS(GLFWwindow * pWindow)
{
	static float previousStamp = 0.0f;
	static int count = 0;

	float currentStamp = glfwGetTime();
	float elapsed = currentStamp - previousStamp;

	if (elapsed > 0.5f)
	{
		previousStamp = currentStamp;
		float fps = count / elapsed;
		int w, h;
		glfwGetFramebufferSize(pWindow,&w,&h);
		char tmp[64];
      	sprintf(tmp,"(%u x %u) - FPS: %.2f",w,h,fps);
		glfwSetWindowTitle(pWindow, tmp);
		count = 0;
	}
	count++;
}

int main(int argc, char **argv)
{
    hipDeviceSetLimit(hipLimitStackSize, 2048);

	GLFWwindow * window;
	if (!glfwInit())
		return -1;

	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);


	window = glfwCreateWindow(800, 600, "Hello World", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}

	glfwMakeContextCurrent(window);
	glfwSwapInterval(0); // disable vsync

	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}

	// Create buffers for rendering
	unsigned int rb, fb;
	glGenRenderbuffers(1, &rb);
	glBindRenderbuffer(GL_RENDERBUFFER, rb);
	glRenderbufferStorage(GL_RENDERBUFFER, GL_RGBA8, 800, 600);
	glBindRenderbuffer(GL_RENDERBUFFER, 0);

	glGenFramebuffers(1, &fb);

	glBindFramebuffer(GL_FRAMEBUFFER, fb);
	glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_RENDERBUFFER, rb);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	hipGraphicsResource_t gr;
	hipGraphicsGLRegisterImage(&gr, rb, GL_RENDERBUFFER, hipGraphicsRegisterFlagsSurfaceLoadStore | hipGraphicsRegisterFlagsWriteDiscard);
	// get cuda array : map + array references + unmap
	hipArray_t ar;
	hipGraphicsMapResources(1, &gr);
	hipGraphicsSubResourceGetMappedArray(&ar, gr, 0, 0);
	hipGraphicsUnmapResources(1, &gr);


	Camera camera;
	Matrix4x4 proj = Matrix4x4::perspective(radians(120), 800.0f/600.0f, 0.1f, 50.0f);
	Matrix4x4 invProj;
	Matrix4x4::invertMatrix(proj, invProj);

	glViewport(0, 0, 800, 600);

	// setup glfw callbacks
	InputHandler handler(window, &camera);
	glfwSetWindowUserPointer(window, &handler);
	glfwSetKeyCallback(window, keyCallback);
	glfwSetCursorPosCallback(window, cursorPosCallback);
	glfwSetMouseButtonCallback(window, mouseButtonCallback);

	// dimensions of the render surface
	dim3 blockDimensions(16, 16);
	dim3 gridDimensions((800+blockDimensions.x-1) / blockDimensions.x, (600+blockDimensions.y-1) / blockDimensions.y);
	// setup gpu memory

	int threadCount = gridDimensions.x*blockDimensions.x*gridDimensions.y*blockDimensions.y;
	Scene * d_scene = importSceneToGPU("scenes/cornell-light-lowpoly2.glb");
	syncAndCheckErrors();

	hiprandState_t * randState;
	hipMalloc(&randState, threadCount * sizeof(hiprandState_t));
	setupRandomState<<<gridDimensions, blockDimensions>>>(randState, time(nullptr));

	while (!glfwWindowShouldClose(window))
	{
		setTitleFPS(window);

		glClear(GL_COLOR_BUFFER_BIT);

		// map cuda array
		hipGraphicsMapResources(1, &gr);

		hipResourceDesc resDesc;
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = ar;
		hipSurfaceObject_t surfObj;
		hipCreateSurfaceObject(&surfObj, &resDesc);

		Matrix4x4 rayTransform = invProj * camera.GetViewMatrix();

		render<<<gridDimensions, blockDimensions>>>(d_scene, 800, 600, 0.1f, camera.Position, rayTransform, surfObj, randState);
		syncAndCheckErrors();

		// unmap cuda array
		hipGraphicsUnmapResources(1, &gr);

		glBindFramebuffer(GL_READ_FRAMEBUFFER, fb);
		glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
		glBlitFramebuffer(0, 0, 800, 600, 0, 600, 800, 0, GL_COLOR_BUFFER_BIT, GL_LINEAR);

		glfwSwapBuffers(window);
		glfwPollEvents();
		handler.handleInputs();
	}

	// cleanup
	destroyScene(d_scene);
	destroyBuffers(rb, fb);
	hipFree(randState);

	glfwTerminate();
	return 0;
}